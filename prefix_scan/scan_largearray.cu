#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "scan_largearray.h"
#include "scan_kernel.cu"

// You can use any other block size you wish.
#define DEFAULT_NUM_ELEMENTS 4096 
#define MAX_RAND 2

typedef float REAL;

// **===-------- Modify the body of this function -----------===**
// You may need to make multiple kernel calls.
void prescanArray(REAL *outArray, REAL *inArray, int num_elements)
{
    REAL* dinArray;
    hipMalloc((void**)&dinArray, num_elements*sizeof(REAL));
    checkError("Malloc dInArray");

    int numThreads = BLOCK_SIZE;
    int numBlocks = (((num_elements+1)/2)+BLOCK_SIZE-1)/BLOCK_SIZE;
    
    REAL* blockSum;
    hipMalloc((void**)&blockSum, numBlocks*sizeof(REAL));
    
    /*----------------------------------------------------------------------------------*/
    // To record timings
    hipEvent_t startEvent_incl, stopEvent_incl;

    hipEventCreate(&startEvent_incl);
    hipEventCreate(&stopEvent_incl);
    // Starting the timer- INCLUSIVE
    hipEventRecord(startEvent_incl, 0);
    /*----------------------------------------------------------------------------------*/

    hipMemcpy(dinArray, inArray, num_elements*sizeof(REAL), hipMemcpyHostToDevice);
    checkError("MemCopy dinArray");
    prefixscan_perblock<<<numBlocks, numThreads>>>(dinArray, blockSum, num_elements);
    checkError("First Kernel");
    
    prefixscan_allblocks<<<numBlocks, numThreads>>>(dinArray, blockSum, num_elements);
    checkError("Second Kernel");
    hipMemcpy(outArray, dinArray, num_elements*sizeof(REAL), hipMemcpyDeviceToHost);
    checkError("Memcopy result");
    
    hipEventRecord(stopEvent_incl, 0);
    hipEventSynchronize(stopEvent_incl);
    // Stopping the timer- INCLUSIVE

    /*----------------------------------------------------------------------------------*/
    float elapsedTime_incl;
    hipEventElapsedTime(&elapsedTime_incl, startEvent_incl, stopEvent_incl);

    printf("Event:Time to complete execution, GPU Inclusive: %f ms\n", elapsedTime_incl);
    // Destroy events
    hipEventDestroy(startEvent_incl);
    hipEventDestroy(stopEvent_incl);
    /*----------------------------------------------------------------------------------*/
    // Free the arrays
    hipFree(dinArray);
    //hipFree(interArray);
    //hipFree(doutArray);
}
// **===-----------------------------------------------------------===**


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C" 
unsigned int compare( const REAL* reference, const REAL* data, 
                     const unsigned int len);
extern "C" 
void computeGold( REAL* reference, REAL* idata, const unsigned int len);

unsigned int cutComparef( REAL *reference, REAL *h_data, int num_elements, REAL err);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    float device_time;
    float host_time;

    int num_elements = 0; // Must support large, non-power-of-2 arrays

    // allocate host memory to store the input data
    unsigned int mem_size = sizeof( REAL) * num_elements;
    REAL* h_data = (REAL*) malloc( mem_size);

    switch(argc-1)
    {      
        case 0:
            num_elements = DEFAULT_NUM_ELEMENTS;
            // allocate host memory to store the input data
            mem_size = sizeof( REAL) * num_elements;
            h_data = (REAL*) malloc( mem_size);

            // initialize the input data on the host
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                int random = rand();
                h_data[i] = floorf(5*(random/(float)RAND_MAX));
                if(random%2)
                    h_data[i] = (-1)*h_data[i];
            }
            break;
        default:
            num_elements = atoi(argv[1]);
            
            // allocate host memory to store the input data
            mem_size = sizeof( REAL) * num_elements;
            h_data = (REAL*) malloc( mem_size);

            // initialize the input data on the host
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
//                h_data[i] = 1.0f;
                h_data[i] = (int)(rand() % MAX_RAND);
            }
        break;  
    }    

    hipEvent_t time_start;
    hipEvent_t time_end;

    hipEventCreate(&time_start);
    hipEventCreate(&time_end);
      
    // compute reference solution
    REAL* reference = (REAL*) malloc( mem_size);  
	// cutStartTimer(timer);
    hipEventRecord(time_start, 0);
    computeGold( reference, h_data, num_elements);
    hipEventRecord(time_end, 0);
    hipEventSynchronize(time_end);
    hipEventElapsedTime(&host_time, time_start, time_end);
	// cutStopTimer(timer);
    printf("\n\n**===-------------------------------------------------===**\n");
    printf("Processing %d elements...\n", num_elements);

    printf("Host CPU Processing time: %f (ms)\n", host_time);


    // allocate device memory input and output arrays
    REAL* d_idata = NULL;
    REAL* d_odata = NULL;

    hipMalloc( (void**) &d_idata, mem_size);
    hipMalloc( (void**) &d_odata, mem_size);
    
    // copy host memory to device input array
    hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice);
    // initialize all the other device arrays to be safe
    hipMemcpy( d_odata, h_data, mem_size, hipMemcpyHostToDevice);

    // **===-------- Allocate data structure here -----------===**
    // preallocBlockSums(num_elements);
    // **===-----------------------------------------------------------===**

    // Run just once to remove startup overhead for more accurate performance 
    // measurement
    //prescanArray(d_odata, d_idata, 16);

    // Run the prescan
    // CUT_SAFE_CALL(cutCreateTimer(&timer));
    // cutStartTimer(timer);

    hipEventRecord(time_start, 0);
    
    // **===-------- Modify the body of this function -----------===**
    prescanArray(d_odata, d_idata, num_elements);
    // **===-----------------------------------------------------------===**
    hipDeviceSynchronize();

    hipEventRecord(time_end, 0);
    hipEventSynchronize(time_end);

    hipEventElapsedTime(&device_time, time_start, time_end);

    hipEventDestroy(time_start);
    hipEventDestroy(time_end);

    // cutStopTimer(timer);
    printf("CUDA Processing time: %g (ms)\n", device_time);
    // device_time = cutGetTimerValue(timer);
    // printf("Speedup: %fX\n", host_time/device_time);

    // **===-------- Deallocate data structure here -----------===**
    // deallocBlockSums();
    // **===-----------------------------------------------------------===**


    // copy result from device to host
    hipMemcpy( h_data, d_odata, sizeof(REAL) * num_elements, 
                               hipMemcpyDeviceToHost);

    // Check if the result is equivalent to the expected soluion
    unsigned int result_regtest = cutComparef( reference, h_data, num_elements, 1e-7);
    printf( "Test %s\n", (0 == result_regtest) ? "FAILED" : "PASSED");

    // cleanup memory
    free( h_data);
    free( reference);
    hipFree( d_odata);
    hipFree( d_idata);
}

unsigned int cutComparef( REAL *reference, REAL *h_data, int num_elements, REAL err) {
    int i;
    int diff_count = 0;
    for (i = 0; i < num_elements; i++) {
        REAL diff = fabs(reference[i] - h_data[i]);
        REAL denominator = 1.f;
        if (denominator < fabs(reference[i])) {
            denominator = fabs(reference[i]);
        }
        if (!(diff / denominator < err)) {
            printf("%d: %f != %f\n", i, reference[i], h_data[i]);
            diff_count ++;
        }
    }
    if (diff_count > 0) {
        printf("Number of difference: %d\n", diff_count);
        return 0;
    } else {
        return 1;
    }
}
